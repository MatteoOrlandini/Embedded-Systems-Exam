/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include gesvdjbatch_example.cpp 
 *   g++ -o gesvdjbatch_example gesvdjbatch_example.o -L/usr/local/cuda/lib64 -lcusolver -lcudart
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            double Areg = A[row + col*lda];
            printf("%s(%d,%d) = %20.16E\n", name, row+1, col+1, Areg);
        }
    }
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    const int m = 3; /* 1 <= m <= 32 */
    const int n = 2; /* 1 <= n <= 32 */
    const int lda = m; /* lda >= m */
    const int ldu = m; /* ldu >= m */
    const int ldv = n; /* ldv >= n */
    const int batchSize = 2;
    const int minmn = (m < n)? m : n; /* min(m,n) */
/*  
 *        |  1  -1  |
 *   A0 = | -1   2  |
 *        |  0   0  |
 *
 *   A0 = U0 * S0 * V0**T
 *   S0 = diag(2.6180, 0.382) 
 *
 *        |  3   4  |
 *   A1 = |  4   7  |
 *        |  0   0  |
 *
 *   A1 = U1 * S1 * V1**T
 *   S1 = diag(9.4721, 0.5279) 
 */
	    double A[lda*n*batchSize]; /* A = [A0 ; A1] */
    double U[ldu*m*batchSize]; /* U = [U0 ; U1] */
    double V[ldv*n*batchSize]; /* V = [V0 ; V1] */
    double S[minmn*batchSize]; /* S = [S0 ; S1] */
    int info[batchSize];       /* info = [info0 ; info1] */

    double *d_A  = NULL; /* lda-by-n-by-batchSize */
    double *d_U  = NULL; /* ldu-by-m-by-batchSize */
    double *d_V  = NULL; /* ldv-by-n-by-batchSize */
    double *d_S  = NULL; /* minmn-by-batchSizee */
    int* d_info  = NULL; /* batchSize */
    int lwork = 0;       /* size of workspace */
    double *d_work = NULL; /* device workspace for gesvdjBatched */

    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const int sort_svd  = 0;   /* don't sort singular values */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */

/* residual and executed_sweeps are not supported on gesvdjBatched */
    double residual = 0;
    int executed_sweeps = 0;

    double *A0 = A;
    double *A1 = A + lda*n; /* Aj is m-by-n */
/*
 *        |  1  -1  |
 *   A0 = | -1   2  |
 *        |  0   0  |
 *   A0 is column-major
 */
    A0[0 + 0*lda] =  1.0;
    A0[1 + 0*lda] = -1.0;
    A0[2 + 0*lda] =  0.0;

    A0[0 + 1*lda] = -1.0;
    A0[1 + 1*lda] =  2.0;
    A0[2 + 1*lda] =  0.0;

/*
 *        |  3   4  |
 *   A1 = |  4   7  |
 *        |  0   0  |
 *   A1 is column-major
 */
    A1[0 + 0*lda] = 3.0;
    A1[1 + 0*lda] = 4.0;
    A1[2 + 0*lda] = 0.0;

    A1[0 + 1*lda] = 4.0;
    A1[1 + 1*lda] = 7.0;
    A1[2 + 1*lda] = 0.0;

    printf("example of gesvdjBatched \n");
    printf("m = %d, n = %d \n", m, n);
    printf("tol = %E, default value is machine zero \n", tol);
    printf("max. sweeps = %d, default value is 100\n", max_sweeps);

    printf("A0 = (matlab base-1)\n");
    printMatrix(m, n, A0, lda, "A0");
    printf("A1 = (matlab base-1)\n");
    printMatrix(m, n, A1, lda, "A1");
    printf("=====\n");
/* step 1: create cusolver handle, bind a stream  */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: configuration of gesvdj */
    status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of tolerance is machine zero */
    status = hipsolverDnXgesvdjSetTolerance(
        gesvdj_params,
        tol);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* default value of max. sweeps is 100 */
    status = hipsolverDnXgesvdjSetMaxSweeps(
        gesvdj_params,
        max_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* disable sorting */
    status = hipsolverDnXgesvdjSetSortEig(
        gesvdj_params,
        sort_svd);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 3: copy A to device */
    cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(double)*lda*n*batchSize);
    cudaStat2 = hipMalloc ((void**)&d_U   , sizeof(double)*ldu*m*batchSize);
    cudaStat3 = hipMalloc ((void**)&d_V   , sizeof(double)*ldv*n*batchSize);
    cudaStat4 = hipMalloc ((void**)&d_S   , sizeof(double)*minmn*batchSize);
    cudaStat5 = hipMalloc ((void**)&d_info, sizeof(int   )*batchSize);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*n*batchSize, hipMemcpyHostToDevice);
    cudaStat2 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    
    /* step 4: query working space of gesvdjBatched */
    status = hipsolverDnDgesvdjBatched_bufferSize(
        cusolverH,
        jobz,
        m,
        n,
        d_A,
        lda,
        d_S,
        d_U,
        ldu,
        d_V,
        ldv,
        &lwork,
        gesvdj_params,
        batchSize
    );
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

/* step 5: compute singular values of A0 and A1 */
    status = hipsolverDnDgesvdjBatched(
        cusolverH,
        jobz,
        m,
        n,
        d_A,
        lda,
        d_S,
        d_U,
        ldu,
        d_V,
        ldv,
        d_work,
        lwork,
        d_info,
        gesvdj_params,
        batchSize
    );
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(U    , d_U   , sizeof(double)*ldu*m*batchSize, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(V    , d_V   , sizeof(double)*ldv*n*batchSize, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(S    , d_S   , sizeof(double)*minmn*batchSize, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info, d_info, sizeof(int) * batchSize       , hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    for(int i = 0 ; i < batchSize ; i++){
        if ( 0 == info[i] ){
            printf("matrix %d: gesvdj converges \n", i);
        }else if ( 0 > info[i] ){
/* only info[0] shows if some input parameter is wrong.
 * If so, the error is CUSOLVER_STATUS_INVALID_VALUE.
 */
            printf("Error: %d-th parameter is wrong \n", -info[i] );
            exit(1);
        }else { /* info = m+1 */
/* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
            printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i] );
        }
    }

/* Step 6: show singular values and singular vectors */
    double *S0 = S;
    double *S1 = S + minmn;
    printf("==== \n");
    for(int i = 0 ; i < minmn ; i++){
        printf("S0(%d) = %20.16E\n", i+1, S0[i]);
    }
    printf("==== \n");
    for(int i = 0 ; i < minmn ; i++){
        printf("S1(%d) = %20.16E\n", i+1, S1[i]);
    }
    printf("==== \n");

    double *U0 = U;
    double *U1 = U + ldu*m; /* Uj is m-by-m */
    printf("U0 = (matlab base-1)\n");
    printMatrix(m, m, U0, ldu, "U0");
    printf("U1 = (matlab base-1)\n");
    printMatrix(m, m, U1, ldu, "U1");

    double *V0 = V;
    double *V1 = V + ldv*n; /* Vj is n-by-n */
    printf("V0 = (matlab base-1)\n");
    printMatrix(n, n, V0, ldv, "V0");
    printf("V1 = (matlab base-1)\n");
    printMatrix(n, n, V1, ldv, "V1");
   /*
 * The folowing two functions do not support batched version.
 * The error CUSOLVER_STATUS_NOT_SUPPORTED is returned. 
 */
    status = hipsolverDnXgesvdjGetSweeps(
        cusolverH,
        gesvdj_params,
        &executed_sweeps);
    assert(HIPSOLVER_STATUS_NOT_SUPPORTED == status);

    status = hipsolverDnXgesvdjGetResidual(
        cusolverH,
        gesvdj_params,
        &residual);
    assert(HIPSOLVER_STATUS_NOT_SUPPORTED == status);

/* free resources */
    if (d_A    ) hipFree(d_A);
    if (d_U    ) hipFree(d_U);
    if (d_V    ) hipFree(d_V);
    if (d_S    ) hipFree(d_S);
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);
    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);

    hipDeviceReset();

    return 0;
}


