#include "hip/hip_runtime.h"
//nvcc OneSidedJacobiSequential.cu svd_one_sided_jacobi_C.cu host_functions.cu -o OneSidedJacobiSequential

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <unistd.h> //for chdir
#include <stdbool.h> //for bool type
#include "host_functions.h" //for host functions
#include "cuda_error_check.h" //for error checking

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

static const float eps = 1e-4;
float * host_B;
float * host_AUX1;
float * cudaB; 

__device__ int sign (float num){
    if (num > 0) return 1;
	if (num < 0) return -1;
	return 0;
}

__global__ void rotate (float * B, int i, int j, int  rows, bool * exit_flag){
	int k = threadIdx.x; 
	__shared__ float alpha, beta, gamm, limit, tao, t, c, s;
	float *pi, *pj;
	if (k < rows) {
		alpha = beta = gamm = 0;
		__syncthreads();
		pi = B + rows * i + k;
		pj = B + rows * j + k;
		atomicAdd(&alpha, *pi * *pi);
		atomicAdd(&beta, *pj * *pj);	
		atomicAdd(&gamm, *pi * *pj);
		__syncthreads();
		if (* exit_flag) {
			//const float limit = fabsf(gamm) / sqrtf(alpha * beta);
			limit = fabsf(gamm) / sqrtf(alpha * beta);
			if (limit > eps) {
				* exit_flag = false;
			}
		}
		//const float tao = (beta - alpha) / (2 * gamm);
		//const float t = sign (tao) / (fabsf(tao) + sqrtf(1 + tao * tao)); 
		//const float c = expf(-0.5f * log1pf(t * t));  // new trick by Giorgio! Better than passing to 64 bits.
		//const float s = c * t;
		tao = (beta - alpha) / (2 * gamm);
		t = sign (tao) / (fabsf(tao) + sqrtf(1 + tao * tao)); 
		c = expf(-0.5f * log1pf(t * t));  // new trick by Giorgio! Better than passing to 64 bits.
		s = c * t;
		const float tmp = *pi;
		*pi = c * tmp - s * *pj;
		*pj = s * tmp + c * *pj;
	}
}

__global__ void computeSingVals (float * B, float * AUX1, int rows, int columns){
	int k = threadIdx.x;
	int j = blockIdx.x;
	__shared__ float t;
	if ((j < columns) && (k < rows)){
		float *pj = B + rows * j + k;
		t = 0;
		atomicAdd(&t, *pj * *pj);
		AUX1[j] = sqrtf(t);
	}
}

int main (int argc, char * argv[])
{
    char input [100], fileName[] = {"Matrix/"};
    chdir("../");
    FILE *fp;
    if (argc == 1){
        do
        {
            printf ("Insert matrix name: \n");
            scanf ("%s", input);
            strcat(fileName, input);
        }
        while(openFile(&fp, fileName, "r") == false);
    }
    else if (argc == 2){
        sprintf(fileName, "Matrix/%s", argv[1]);
        if (openFile(&fp, fileName, "r") == false)
        	exit(1);
	} 
    float * matrix;
    int rows, columns;
    fillRowMajorOrderMatrix(&fp, &matrix, &rows, &columns);
    fclose(fp);
    // Column order matrix
    createColumnMajorOrderMatrix(&host_B, matrix, rows, columns);
    // initialize host_AUX1 array to zero
    initializeArray (&host_AUX1, columns);
    // Open new file to store the singular values
    sprintf(fileName, "SingularValues/CudaHost/Singular Values Cuda Host %dx%d.txt", rows, columns);
    openFile(&fp, fileName, "w");
    // cuda events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
    //compute one sided jacobi
    int iterations = svd_one_sided_jacobi_C(rows, columns);
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds_host = 0;
	hipEventElapsedTime(&milliseconds_host, start, stop);
	hipEventDestroy (start); 
	hipEventDestroy (stop); 
    printf("Iterations on host: %d \n", iterations);
    descentOrdering(host_AUX1, columns);
    //print array and save on file
	printAndSaveArray (&fp, host_AUX1, columns);
    fclose(fp);
	
	/****************************************************************************************************/
	/****************************************************************************************************/
	/****************************************************************************************************/
	/****************************************************************************************************/
	free(host_B);
	createColumnMajorOrderMatrix(&host_B, matrix, rows, columns);
	
	float *AUX1, * B;	
	bool * exit_flag;

	CudaSafeCall(hipMalloc( (void**)&B, rows*columns*sizeof(float) ));
	CudaSafeCall(hipMalloc( (void**)&exit_flag, sizeof(bool) ));
	CudaSafeCall(hipMalloc( (void**)&AUX1, columns * sizeof(float) ));

	CudaSafeCall(hipMemcpy( B, host_B, rows * columns * sizeof(float), hipMemcpyHostToDevice));

	//printf ("MATRIX ALLOCATED ON DEVICE \n");
	//dim3 block(rows, 0, 0);
	//dim3 grid(1, 0, 0);
	int iter = 0;
	bool host_exit_flag = false;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	while (!host_exit_flag) {
	++iter;
	host_exit_flag = true;
	CudaSafeCall(hipMemcpy( exit_flag, &host_exit_flag, sizeof(bool), hipMemcpyHostToDevice));
	for (int j = columns - 1; j >= 1; --j)
		for (int i = j - 1; i >= 0; --i) {	
			rotate<<<1, rows>>> (B, i, j, rows, exit_flag);
			CudaCheckError();
		}
	CudaSafeCall(hipMemcpy( &host_exit_flag, exit_flag, sizeof(bool), hipMemcpyDeviceToHost));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds_device = 0;
	hipEventElapsedTime(&milliseconds_device, start, stop);
	hipEventDestroy (start); 
	hipEventDestroy (stop);
	//Open new file to store the singular values on device
	sprintf(fileName, "SingularValues/CudaDevice/OneSidedSequential/Singular Values Cuda Device %dx%d.txt", rows, columns);
	openFile(&fp, fileName, "w");

	printf ("Iterations on device: %d\n", iter);

	// calculate singular values
	computeSingVals<<<columns, rows>>> (B, AUX1, rows, columns);
	CudaCheckError();
	CudaSafeCall(hipMemcpy( host_AUX1, AUX1, columns * sizeof(float),  hipMemcpyDeviceToHost));

	descentOrdering(host_AUX1, columns);
	printAndSaveArray (&fp, host_AUX1, columns);
	fclose(fp);
	//printf("SINGULAR VALUES STORED TO FILE \n");

	// free the memory allocated on the CPU 
    free(host_B);
	free(host_AUX1);

	//printf ("MEMORY ON HOST DEALLOCATED \n");

	// free the memory allocated on the GPU
	hipFree(B);
	hipFree(exit_flag);
	hipFree(AUX1);

	//printf ("MEMORY ON DEVICE DEALLOCATED \n");
	
	printf ("Time on host: %f ms\n", milliseconds_host);
	printf ("Time on device: %f ms\n", milliseconds_device);

	sprintf(fileName, "Time/CudaHost/Time %dx%d.txt", rows, columns);
	openFile(&fp, fileName, "w");
	fprintf(fp, "%f", milliseconds_host);	
	fclose(fp);
	
	sprintf(fileName, "Time/CudaDevice/OneSidedSequential/Time %dx%d.txt", rows, columns);
	openFile(&fp, fileName, "w");
	fprintf(fp, "%f", milliseconds_device);	
	fclose(fp);
		
	hipDeviceReset ( ); 
	return 0;
}


