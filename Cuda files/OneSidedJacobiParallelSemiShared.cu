#include "hip/hip_runtime.h"
//nvcc OneSidedJacobiParallelSemiShared.cu svd_one_sided_jacobi_C.cu host_functions.cu -o OneSidedJacobiParallelSemiShared

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <unistd.h> //for chdir
#include <stdbool.h> //for bool type
#include "host_functions.h" //for host functions
#include "cuda_error_check.h" //for error checking

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

static const float eps = 1e-4;
float * host_B;
float * host_AUX1;

__device__ int sign (float num){
	if (num > 0) return 1;
	if (num < 0) return -1;
	return 0;
}


__global__ void scheduling (int *v1, int *v2, int cols){
	int tmp = v2[0];
	for (int i = 0; i < (cols/2) - 1; i++)
		v2[i] = v2[i+1];	
	v2[cols/2 - 1] = v1[cols/2 - 1];
	for (int i = (cols/2) -1; i > 1; i--)
		v1[i] = v1[i-1];	
	v1[1] = tmp;
}


__global__ void round (float *B, int *v1, int *v2, int cols, int rows, bool * exit_flag) {
	int blockId = blockIdx.x; //max(blockId) = (cols/2) - 1
	int threadId = threadIdx.x; //max(blockId) = rows - 1
	float * pi, *pj;
	__shared__ float alpha, beta, gamm, limit, tao, t, c, s;
	__shared__ int i, j;
	if ((blockId < cols/2) && (threadId < rows)){
		i = *(v1 + blockId);
		j = *(v2 + blockId);
		pi = B + rows * i + threadId;
		pj = B + rows * j + threadId;
		alpha = beta = gamm = 0;
		__syncthreads();
		atomicAdd(&alpha, *pi * *pi);
		atomicAdd(&beta, *pj * *pj);	
		atomicAdd(&gamm, *pi * *pj);
		__syncthreads();
		if ( *exit_flag) {
			limit = fabsf(gamm) / sqrtf(alpha * beta);
			if (limit > eps){
				*exit_flag = false;
			}
		} 
		tao = (beta - alpha) / (2 * gamm);
		t = sign (tao) / (fabsf(tao) + sqrtf(1 + tao * tao)); 
		c = expf(-0.5f * log1pf(t * t));
		s = c * t;
		const float tmp = *pi;
		*pi = c * tmp - s * *pj;
		*pj = s * tmp + c * *pj;
	}
}

__global__ void computeSingVals (float * B, float * AUX1, int rows, int columns){
	int k = threadIdx.x; //max(k)=rows-1
	int j = blockIdx.x; //max(j)=columns-1
	__shared__ float t;
	if ((j < columns) && (k < rows)){
		float *pj = B + rows * j + k;
		t = 0;
		__syncthreads();
		atomicAdd(&t, *pj * *pj);
		__syncthreads();
		AUX1[j] = sqrtf(t);
	}
}

int main (int argc, char * argv[]) {
	char input [100], fileName[] = {"Matrix/"};
	chdir("../");
	FILE *fp;
	if (argc == 1){
	    do
	    {
	        printf ("Insert matrix name: \n");
	        scanf ("%s", input);
	        strcat(fileName, input);
	    }
	    while(openFile(&fp, fileName, "r") == false);
	}
	else if (argc == 2){
	    sprintf(fileName, "Matrix/%s", argv[1]);
	    if (openFile(&fp, fileName, "r") == false)
	    	exit(1);
	} 
	float * matrix;
	int rows, cols;
	fillRowMajorOrderMatrix(&fp, &matrix, &rows, &cols);
	fclose(fp);
	// Column order matrix
	createColumnMajorOrderMatrix(&host_B, matrix, rows, cols);
	// initialize host_AUX1 array to zero
	initializeArray (&host_AUX1, cols);
	// Open new file to store the singular values
	sprintf(fileName, "SingularValues/CudaHost/Singular Values Cuda Host %dx%d.txt", rows, cols);
	openFile(&fp, fileName, "w");
	// cuda events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	//compute one sided jacobi
	int iterations = svd_one_sided_jacobi_C(rows, cols);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds_host = 0;
	hipEventElapsedTime(&milliseconds_host, start, stop);
	hipEventDestroy (start); 
	hipEventDestroy (stop); 
	printf("Iterations on host: %d \n", iterations);
	descentOrdering(host_AUX1, cols);
	//print array and save on file
	printAndSaveArray (&fp, host_AUX1, cols);
	fclose(fp);

	/****************************************************************************************************/
	/****************************************************************************************************/
	/****************************************************************************************************/
	/****************************************************************************************************/
	free(host_B);
	createColumnMajorOrderMatrix(&host_B, matrix, rows, cols);
	/*
	for (int i = 0; i < rows; i++){
		for (int j = 0; j < cols; j++){
			printf ("B[%d]:%f\t", i*cols+j, host_B[i*cols+j]);
		}
		printf ("\n");
	}
	*/

	float * dev_AUX1, * dev_B;
	int * dev_v1, * dev_v2;	
	bool * dev_exit_flag;

	CudaSafeCall(hipMalloc( (void**)&dev_B, rows*cols*sizeof(float) ));
	CudaSafeCall(hipMalloc( (void**)&dev_exit_flag, sizeof(bool) ));
	CudaSafeCall(hipMalloc( (void**)&dev_AUX1, cols * sizeof(float) ));
	CudaSafeCall(hipMalloc( (void**)&dev_v1, (cols/2) * sizeof(float) ));
	CudaSafeCall(hipMalloc( (void**)&dev_v2, (cols/2) * sizeof(float) ));
	/***********************************/
	int host_v1[cols/2], host_v2[cols/2];
	for (int i = 0; i < cols/2; i++) {
		host_v1[i] = i*2;
		host_v2[i] = i*2 + 1;
	}
	/***********************************/
	CudaSafeCall(hipMemcpy( dev_B, host_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy( dev_v1, host_v1, (cols/2) * sizeof(float), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy( dev_v2, host_v2, (cols/2) * sizeof(float), hipMemcpyHostToDevice));

	//printf ("MATRIX ALLOCATED ON DEVICE \n");
	int iter = 0;
	bool host_exit_flag = false;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	while(!host_exit_flag) {
		++iter;
		host_exit_flag = true; //rimuovendo si perde accuratezza
		CudaSafeCall(hipMemcpy( dev_exit_flag, &host_exit_flag, sizeof(bool), hipMemcpyHostToDevice));
		for(int set = 0; set < cols; set++) { //rimuovendo il for si perde un po' di accuratezza ma dimuiscono le iterazioni
			scheduling<<<1,1>>> (dev_v1, dev_v2, cols);
			CudaCheckError();
			round <<<cols/2, rows>>> (dev_B, dev_v1, dev_v2, cols, rows, dev_exit_flag);
			CudaCheckError();		
		}
		CudaSafeCall(hipMemcpy( &host_exit_flag, dev_exit_flag, sizeof(bool), hipMemcpyDeviceToHost));
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds_device = 0;
	hipEventElapsedTime(&milliseconds_device, start, stop);
	// open new file to store the singular values on device
	sprintf(fileName, "SingularValues/CudaDevice/OneSidedParallelSemiShared/Singular Values Cuda Device %dx%d.txt", rows, cols);
	openFile(&fp, fileName, "w");

	printf ("Iterations on device: %d\n", iter);

	// calculate singular values
	computeSingVals<<<cols, rows>>> (dev_B, dev_AUX1, rows, cols);
	CudaCheckError();
	CudaSafeCall(hipMemcpy( host_AUX1, dev_AUX1, cols * sizeof(float),  hipMemcpyDeviceToHost));

	descentOrdering(host_AUX1, cols);
	printAndSaveArray (&fp, host_AUX1, cols);
	fclose(fp);
	//printf("SINGULAR VALUES STORED TO FILE \n");

	// free the memory allocated on host
	free(host_B);
	free(host_AUX1);

	//printf ("MEMORY ON HOST DEALLOCATED \n");

	// free the memory allocated on device
	hipFree(dev_B);
	hipFree(dev_exit_flag);
	hipFree(dev_AUX1);
	hipFree(dev_v1);
	hipFree(dev_v2);

	//printf ("MEMORY ON DEVICE DEALLOCATED \n");

	printf ("Time on host: %f ms\n", milliseconds_host);
	printf ("Time on device: %f ms\n", milliseconds_device);

	sprintf(fileName, "Time/CudaHost/Time %dx%d.txt", rows, cols);
	openFile(&fp, fileName, "w");
	fprintf(fp, "%f", milliseconds_host);	
	fclose(fp);

	sprintf(fileName, "Time/CudaDevice/OneSidedParallelSemiShared/Time %dx%d.txt", rows, cols);
	openFile(&fp, fileName, "w");
	fprintf(fp, "%f", milliseconds_device);	
	fclose(fp);

	hipDeviceReset ( ); 

	return 0;
}
		

