#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
	cudaProfilerInitialize ( "config.txt", "output.txt", cudaCSV);
	hipProfilerStart();
    cuda_hello<<<1,1>>>(); 
    hipProfilerStop();
    return 0;
}

